#include "hip/hip_runtime.h"
// Copyright 2022 Risc0, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "fpext.h"

__global__ void combos_prepare(FpExt* combos,
                               const FpExt* coeffU,
                               const uint32_t regsCount,
                               const uint32_t* regSizes,
                               const uint32_t* regComboIds,
                               const uint32_t cycles,
                               const FpExt* mix,
                               const uint32_t checkSize,
                               const uint32_t comboCount) {
  uint gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid > 1) {
    return;
  }

  uint32_t pos = 0;
  FpExt cur(1);

  // Subtract the U coeffs from the combos
  for (uint32_t i = 0; i < regsCount; i++) {
    uint32_t regSize = regSizes[i];
    uint32_t regComboId = regComboIds[i];
    for (uint32_t j = 0; j < regSize; j++) {
      combos[cycles * regComboId + j] -= cur * coeffU[pos + j];
    }
    cur *= *mix;
    pos += regSize;
  }

  // Subtract the final 'check' coefficients
  for (uint32_t i = 0; i < checkSize; i++) {
    combos[cycles * comboCount] -= cur * coeffU[pos];
    pos++;
    cur *= *mix;
  }
}
