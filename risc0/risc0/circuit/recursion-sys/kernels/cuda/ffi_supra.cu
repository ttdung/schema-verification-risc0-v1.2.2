// Copyright 2024 RISC Zero, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "hip/hip_runtime.h"
#include "supra/fp.h"

#include "kernels.h"

using namespace risc0::circuit::recursion;

extern "C" {

const char* risc0_circuit_recursion_cuda_eval_check(Fp* check,
                                                    const Fp* ctrl,
                                                    const Fp* data,
                                                    const Fp* accum,
                                                    const Fp* mix,
                                                    const Fp* out,
                                                    const Fp& rou,
                                                    uint32_t po2,
                                                    uint32_t domain,
                                                    const FpExt* poly_mix_pows) {
  hipMemcpyToSymbol(HIP_SYMBOL(poly_mix), poly_mix_pows, sizeof(poly_mix));
  return launchKernel(eval_check, domain, 0, check, ctrl, data, accum, mix, out, rou, po2, domain);
}

} // extern "C"
